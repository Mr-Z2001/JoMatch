#include "hip/hip_runtime.h"
#include "gpu_match.cuh"
#include "cuda_helpers.h"
#include "order.h"
#include "join.cuh"
#include "memManag.cuh"
#include "res_table.hpp"

#include <hip/hip_cooperative_groups.h>

#include <ctime>
#include <chrono>

__global__ void
warmup()
{
  int a = IDX * 1;
}

__device__ void
intersect(Arg_t *__restrict__ arg)
{
  __shared__ offtype off[WARP_PER_BLOCK];
  __shared__ offtype off_ed[WARP_PER_BLOCK];
  __shared__ bool found[WARP_PER_BLOCK][WARP_SIZE];
  __shared__ vtype v[WARP_PER_BLOCK][WARP_SIZE];

  __shared__ int cnt[WARP_PER_BLOCK];

  if (LID == 0)
  {
    cnt[WID] = 0;
    off[WID] = 0;
    off_ed[WID] = arg->set1_size;
  }
  __syncwarp();
  // vtype v;
  // bool found = false;

  while (off[WID] < off_ed[WID])
  {
    // off[WID] + lid = off[WID] + LID;
    v[WID][LID] = UINT32_MAX;
    if (off[WID] + LID < off_ed[WID])
      v[WID][LID] = arg->set1[off[WID] + LID];
    // else
    // v[WID][LID] = UINT32_MAX;
    __syncwarp();
    found[WID][LID] = false;
    if (v[WID][LID] != UINT32_MAX)
    {
      int res = lower_bound(arg->set2, arg->set2_size, v[WID][LID]);
      if (res != UINT32_MAX && arg->set2[res] == v[WID][LID])
        found[WID][LID] = true;
      // #pragma unroll 8
      //       for (int i = 0; i < arg->set2_size; ++i)
      //       {
      //         // found[WID][LID] |= (arg->set2[i] == v[WID][LID]);
      //         if (arg->set2[i] == v[WID][LID])
      //         {
      //           found[WID][LID] = true;
      //           break;
      //         }
      //         else if (arg->set2[i] > v[WID][LID])
      //         {
      //           break;
      //         }
      //       }
    }
    __syncwarp();
    if (found[WID][LID])
    {
      int mask = __activemask();
      int size = __popc(mask);
      int rank = __popc(mask & (FULL_MASK >> (31 - LID))) - 1;
      int pos = cnt[WID];
      // if (pos + rank > C_NUM_CAN_UB)
      // printf("pos + rank = %d\n", pos + rank);
      arg->res[pos + rank] = v[WID][LID];

      if (rank == 0)
        cnt[WID] += size;
    }
    __syncwarp();
    if (LID == 0)
      off[WID] += warpSize;
    __syncwarp();
  }
  __syncwarp();
  if (LID == 0)
    arg->res_size[0] = cnt[WID];
  __syncwarp();
}

__device__ void
get_new_v(
    degtype *__restrict__ d_degs_, offtype *__restrict__ d_offsets_, vtype *__restrict__ d_nbrs_,
    CallStack *__restrict__ stk, int *__restrict__ cur,
    StealingArgs *__restrict__ stealing_args, long long &start_clk,
    OrderGPU *__restrict__ order_obj,

    int start_level, // matched.

    uint32_t *__restrict__ compact_encodings_,
    int num_blocks,

    vtype *__restrict__ initial_task_table_, numtype num_initial_task_table_rows,

    bool *__restrict__ ret, vtype *__restrict__ cur_res, int *order_id)
{
  if (stk->level == start_level)
  {
    __shared__ int next_pos[WARP_PER_BLOCK];
    // __shared__ vtype u[WARP_PER_BLOCK][WARP_SIZE];
    // int next_pos;
    if (LID == 0)
      next_pos[WID] = atomicAdd(cur, 1);
    __syncwarp();
    // next_pos = __shfl_sync(FULL_MASK, next_pos, 0);

    if (next_pos[WID] >= num_initial_task_table_rows)
    {
      if (LID == 0)
        *ret = false;
      __syncwarp();
      return;
    }

    if (LID == 0)
    {
      *order_id = initial_task_table_[next_pos[WID] * (start_level + 1 + 1) + 0];
      // stk->iter[LID] = 0;
    }
    // int l = LID; // one lane -- one level
    else if (LID <= start_level + 1)
    {
      // u[WID][LID] = order_obj->v_orders_[order_id * C_NUM_VQ + l];
      // u[WID][LID] = v_order[l];
      stk->iter[LID - 1] = 0;
      cur_res[LID - 1] = initial_task_table_[next_pos[WID] * (start_level + 1 + 1) + LID];
      // cur_res[u[WID][LID]] = initial_task_table_[next_pos[WID] * C_NUM_VQ + u[WID][LID]];
      // stk->candidates_[l * C_NUM_CAN_UB + 0] = initial_task_table_[next_pos * C_NUM_VQ + u];
      stk->num_candidates_[LID - 1] = 1;
    }
    __syncwarp();
  }
  else // level < C_NUM_VQ - 1
  {
    __shared__ vtype u[WARP_PER_BLOCK];
    __shared__ int cnt[WARP_PER_BLOCK];
    __shared__ Arg_t arg[WARP_PER_BLOCK];
    __shared__ vtype mapped_vs_[WARP_PER_BLOCK][WARP_SIZE];

    __shared__ int min_i[WARP_PER_BLOCK];
    __shared__ int min_nbrs[WARP_PER_BLOCK];
    __shared__ bool flag[WARP_PER_BLOCK][MAX_VQ];

    if (LID < C_NUM_VQ)
      flag[WID][LID] = false;
    __syncwarp();

    if (LID == 0)
    {
      // printf("order_id: %d\n", *order_id);
      u[WID] = order_obj->v_orders_[(*order_id) * C_NUM_VQ + stk->level];
    }
    __syncwarp();
    // u[WID] = v_order[stk->level];
    // vtype u = v_order[stk->level];

    // if (LID == 0)
    //   *cnt = 0;
    // __syncwarp();

    // if (LID == 0)
    // printf("num_bn: %d\n", order_obj->num_backward_neighbors_[u]);
    // __syncwarp();

    if (LID < order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]])
    {
      vtype u_back = order_obj->backward_neighbors_[(*order_id) * C_NUM_VQ * C_NUM_VQ + u[WID] * C_NUM_VQ + LID];
      // int l_back = order_obj->u2l_[u_back];
      // vtype v_back = cur_res[u_back];
      // vtype v_back = stk->candidates_[l_back * C_NUM_CAN_UB + stk->iter[l_back]];
      // mapped_vs_[WID][LID] = cur_res[u_back];
      mapped_vs_[WID][LID] = cur_res[order_obj->u2ls_[(*order_id) * C_NUM_VQ + u_back]];
    }
    __syncwarp();

    if (LID == 0)
    {
      if (order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]] == 1)
      {
        min_i[WID] = 0;
      }
      else
      {
        min_i[WID] = 0;
        min_nbrs[WID] = d_degs_[mapped_vs_[WID][0]];
        for (int i = 1; i < order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]]; ++i)
        {
          if (d_degs_[mapped_vs_[WID][i]] < min_nbrs[WID])
          {
            min_i[WID] = i;
            min_nbrs[WID] = d_degs_[mapped_vs_[WID][i]];
          }
        }
      }
    }
    __syncwarp();

    if (LID == 0)
    {
      // printf("mapped_vs_[0] = %d\n", mapped_vs_[0]);
      arg[WID].res = d_nbrs_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
      arg[WID].res_size = stk->num_candidates_ + stk->level;
      arg[WID].res_size[0] = d_degs_[mapped_vs_[WID][min_i[WID]]];
      flag[WID][min_i[WID]] = true;
    }
    __syncwarp();
    for (int i = 1; i < order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]]; ++i)
    {
      if (LID == 0)
      {
        min_nbrs[WID] = UINT32_MAX;
        for (int j = 0; j < order_obj->num_backward_neighbors_[(*order_id) * C_NUM_VQ + u[WID]]; ++j)
        {
          if (flag[WID][j])
            continue;
          if (d_degs_[mapped_vs_[WID][j]] < min_nbrs[WID])
          {
            min_i[WID] = j;
            min_nbrs[WID] = d_degs_[mapped_vs_[WID][j]];
          }
        }

        arg[WID].set1 = arg[WID].res;
        arg[WID].set1_size = arg[WID].res_size[0];
        arg[WID].set2 = d_nbrs_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
        arg[WID].set2_size = d_degs_[mapped_vs_[WID][min_i[WID]]];
        // arg[WID].res = can + (stk->level - start_level - 1) * C_NUM_CAN_UB;
        arg[WID].res = stk->candidates_ + stk->level * C_NUM_CAN_UB;
        arg[WID].res_size = stk->num_candidates_ + stk->level;
        flag[WID][min_i[WID]] = true;
      }
      __syncwarp();
      intersect(&arg[WID]);
    }

    if (LID == 0)
      cnt[WID] = 0;
    __syncwarp();

    __shared__ offtype off[WARP_PER_BLOCK];
    __shared__ offtype off_ed[WARP_PER_BLOCK];
    __shared__ bool dup[WARP_PER_BLOCK][WARP_SIZE];
    __shared__ vtype v[WARP_PER_BLOCK][WARP_SIZE];

    if (LID == 0)
    {
      off[WID] = 0;
      off_ed[WID] = arg[WID].res_size[0];
    }
    __syncwarp();
    // vtype v;
    // bool dup = false;
    while (off[WID] < off_ed[WID])
    {
      int my_off = off[WID] + LID;
      if (my_off < off_ed[WID])
        v[WID][LID] = arg[WID].res[my_off];
      else
        v[WID][LID] = UINT32_MAX;
      __syncwarp();
      dup[WID][LID] = false;
      if (v[WID][LID] != UINT32_MAX)
        for (int l = 0; l < stk->level; ++l)
        {
          if (v[WID][LID] == cur_res[l])
          // if (v[WID][LID] == cur_res[v_order[l]])
          {
            dup[WID][LID] = true;
            break;
          }
          // if (v[WID][LID] == stk->candidates_[l * C_NUM_CAN_UB + stk->iter[l]])
          // {
          //   dup = true;
          //   break;
          // }
        }
      __syncwarp();
      if (v[WID][LID] != UINT32_MAX &&
          !dup[WID][LID] &&
          (compact_encodings_[u[WID] * C_COL_LEN + v[WID][LID] / BLK_SIZE] & (1 << (v[WID][LID] % BLK_SIZE))))
      {
        int mask = __activemask();
        int size = __popc(mask);
        int rank = __popc(mask & (FULL_MASK >> (31 - LID))) - 1;
        int my_pos = cnt[WID] + rank;
        // can[(stk->level - start_level - 1) * C_NUM_CAN_UB + my_pos] = v[WID][LID];
        stk->candidates_[stk->level * C_NUM_CAN_UB + my_pos] = v[WID][LID];
        if (rank == 0)
          cnt[WID] += size;
      }
      __syncwarp();
      if (LID == 0)
        off[WID] += warpSize;
      __syncwarp();
    }
    __syncwarp();
    if (LID == 0)
    {
      if (cnt[WID] == 0)
        *ret = false;
      else
      {
        stk->iter[stk->level] = 0;
        stk->num_candidates_[stk->level] = cnt[WID];
        // cur_res[u[WID]] = can[(stk->level - start_level - 1) * C_NUM_CAN_UB + 0];
        // cur_res[u[WID]] = stk->candidates_[stk->level * C_NUM_CAN_UB + 0];
        cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + 0];
      }
    }
    __syncwarp();
  }
}

__device__ void
steal(
    CallStack *__restrict__ stk, StealingArgs *__restrict__ stealing_args, bool *__restrict__ __restrict__ ret, long long &start_clk, vtype *v_order,
    int *queue_arr_,
    vtype *__restrict__ cur_res, int *order_id)
{
  __shared__ bool flag[WARP_PER_BLOCK];

  if (LID == 0)
  {
    stk->stealed_task = false;
    flag[WID] = stealing_args->queue->dequeue(queue_arr_, C_STOP_LEVEL + 1);
  }
  __syncwarp();

  if (flag[WID])
  {
    if (LID < C_STOP_LEVEL + 1 && LID > 0)
    {
      if (queue_arr_[LID] != DeletionMarker<int>::val - 1)
      {
        stk->iter[LID - 1] = 0;
        stk->num_candidates_[LID - 1] = 1;
        // stk->candidates_[LID * C_NUM_CAN_UB + 0] = queue_arr_[LID];
        // cur_res[v_order[LID]] = queue_arr_[LID];
        cur_res[LID - 1] = queue_arr_[LID];
        atomicMax(&stk->level, LID - 1);
      }
    }
    else if (LID == 0)
    {
      *order_id = queue_arr_[0];
    }
    __syncwarp();
    if (LID == 0)
    {
      stk->iter[stk->level + 1] = 0;
      stk->num_candidates_[stk->level + 1] = 0;
      stk->stealed_task = true;
    }
    __syncwarp();
  }
  else
  {
    if (LID == 0)
      *ret = false;
    __syncwarp();
  }
  if (LID == 0)
    start_clk = clock64();
  __syncwarp();
  // start_clk = __shfl_sync(FULL_MASK, start_clk, 0);

  // if (LID == 0)
  // {
  //   stk->stealed_task = false;
  //   bool flag = stealing_args->queue->dequeue(queue_arr_, C_STOP_LEVEL);
  //   if (flag)
  //   {
  //     for (int i = 0; i < C_STOP_LEVEL; ++i)
  //     {
  //       int val = queue_arr_[i];
  //       if (val != DeletionMarker<int>::val - 1)
  //       {
  //         stk->iter[i] = 0;
  //         stk->num_candidates_[i] = 1;
  //         // stk->candidates_[i * C_NUM_CAN_UB + 0] = val;
  //         cur_res[v_order[i]] = val;
  //         stk->level = i;
  //       }
  //       else
  //       {
  //         stk->iter[i] = 0;
  //         stk->num_candidates_[i] = 0;
  //         break;
  //       }
  //     }
  //     stk->stealed_task = true;
  //   }
  //   else
  //   {
  //     *ret = false;
  //   }
  // }
  // __syncwarp();
  // if (LID == 0)
  //   start_clk = clock64();
  // start_clk = __shfl_sync(FULL_MASK, start_clk, 0);
}

__device__ void
match(
    degtype *__restrict__ d_degs_, offtype *__restrict__ d_offsets_, vtype *__restrict__ d_nbrs_,
    CallStack *__restrict__ stk, int *__restrict__ cur, unsigned long long *__restrict__ count,
    StealingArgs *__restrict__ stealing_args, long long &start_clk,
    OrderGPU *__restrict__ order_obj, int start_level, int *__restrict__ queue_arr_,
    uint32_t *__restrict__ compact_encodings_, int num_blocks,
    vtype *__restrict__ initial_task_table_, numtype num_initial_task_table_rows,

    bool *ret, vtype *cur_res)
{
  // uint32_t &level = stk->level;
  // can = candidates[wid]

  __shared__ int s_order_id[WARP_PER_BLOCK];
  if (LID == 0)
    s_order_id[WID] = -1;
  __syncwarp();

  while (true)
  {
    if (stk->level == start_level)
    {
      if (LID == 0)
        *ret = true;
      __syncwarp();

      // steal(stk, stealing_args, ret, start_clk, v_order, queue_arr_, cur_res);
      steal(stk, stealing_args, ret, start_clk, order_obj->v_orders_, queue_arr_, cur_res, &s_order_id[WID]);
      if (*ret == false) // nothing to steal
      {
        if (LID == 0)
          *ret = true;
        __syncwarp();
        get_new_v(
            d_degs_, d_offsets_, d_nbrs_,
            stk, cur, stealing_args, start_clk,
            order_obj, start_level,
            compact_encodings_, num_blocks,
            initial_task_table_, num_initial_task_table_rows,
            ret, cur_res, &s_order_id[WID]);
        __syncwarp();
      }
      else
      {
        if (LID == 0)
        {
          *ret = true;
        }
        __syncwarp();
      }
      if (*ret == false) // no more candidates
      {
        break;
      }
      else
      {
        if (LID == 0)
        {
          stk->level++;
          stk->iter[stk->level] = 0;
          stk->num_candidates_[stk->level] = 0;
        }
        __syncwarp();
      }
    }
    else if (stk->level == C_NUM_VQ - 1) // final, intersect and return
    {
      __shared__ Arg_t arg[WARP_PER_BLOCK];
      __shared__ vtype mapped_vs_[WARP_PER_BLOCK][WARP_SIZE];
      __shared__ int cnt[WARP_PER_BLOCK];

      __shared__ int min_i[WARP_PER_BLOCK];
      __shared__ int min_nbrs[WARP_PER_BLOCK];
      __shared__ bool flag[WARP_PER_BLOCK][MAX_VQ];

      if (LID < C_NUM_VQ)
        flag[WID][LID] = false;
      __syncwarp();

      if (LID == 0)
      {
        if (s_order_id[WID] == -1)
        {
        }
      }

      vtype u = order_obj->v_orders_[s_order_id[WID] * C_NUM_VQ + stk->level];
      // vtype u = v_order[stk->level];

      // if (LID == 0)
      // {
      //   printf("s_order_id[WID]: %d \n", s_order_id[WID]);
      // }
      // __syncwarp();
      if (LID < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u])
      {
        vtype u_back = order_obj->backward_neighbors_[s_order_id[WID] * C_NUM_VQ * C_NUM_VQ + u * C_NUM_VQ + LID];
        // vtype u_back = order_obj->backward_neighbors_[u * C_NUM_VQ + LID];
        // int l_back = order_obj->u2l_[u_back];
        // vtype v_back = cur_res[u_back];
        // vtype v_back = stk->candidates_[l_back * C_NUM_CAN_UB + stk->iter[l_back]];
        // mapped_vs_[WID][LID] = cur_res[u_back];
        mapped_vs_[WID][LID] = cur_res[order_obj->u2ls_[s_order_id[WID] * C_NUM_VQ + u_back]];
      }
      __syncwarp();

      if (LID == 0)
      {
        if (order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u] == 1)
        {
          min_i[WID] = 0;
        }
        else
        {
          min_i[WID] = 0;
          min_nbrs[WID] = d_degs_[mapped_vs_[WID][0]];
          for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u]; ++i)
          {
            if (d_degs_[mapped_vs_[WID][i]] < min_nbrs[WID])
            {
              min_i[WID] = i;
              min_nbrs[WID] = d_degs_[mapped_vs_[WID][i]];
            }
          }
        }
      }
      __syncwarp();

      if (LID == 0)
      {
        arg[WID].res = d_nbrs_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
        arg[WID].res_size = stk->num_candidates_ + stk->level;
        arg[WID].res_size[0] = d_degs_[mapped_vs_[WID][min_i[WID]]];
        flag[WID][min_i[WID]] = true;
      }
      __syncwarp();
      for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u]; ++i)
      {
        if (LID == 0)
        {
          min_nbrs[WID] = UINT32_MAX;
          for (int j = 0; j < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + u]; ++j)
          {
            if (flag[WID][j])
              continue;
            if (d_degs_[mapped_vs_[WID][j]] < min_nbrs[WID])
            {
              min_i[WID] = j;
              min_nbrs[WID] = d_degs_[mapped_vs_[WID][j]];
            }
          }

          arg[WID].set1 = arg[WID].res;
          arg[WID].set1_size = arg[WID].res_size[0];
          arg[WID].set2 = d_nbrs_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
          arg[WID].set2_size = d_degs_[mapped_vs_[WID][min_i[WID]]];
          // arg[WID].res = can_wid + (stk->level - start_level - 1) * C_NUM_CAN_UB;
          arg[WID].res = stk->candidates_ + stk->level * C_NUM_CAN_UB;
          arg[WID].res_size = stk->num_candidates_ + stk->level;
          flag[WID][min_i[WID]] = true;
        }
        __syncwarp();
        intersect(&arg[WID]);
      }

      if (LID == 0)
        cnt[WID] = 0;
      __syncwarp();
      // int pos = d_enc_pos_u_[u];

      __shared__ offtype off_ed[WARP_PER_BLOCK];
      __shared__ offtype off[WARP_PER_BLOCK];
      __shared__ bool dup[WARP_PER_BLOCK][WARP_SIZE];
      __shared__ vtype v[WARP_PER_BLOCK][WARP_SIZE];

      off[WID] = 0;
      off_ed[WID] = arg[WID].res_size[0];
      // vtype v;
      dup[WID][LID] = false;
      while (off[WID] < off_ed[WID])
      {
        if (off[WID] + LID < off_ed[WID])
          v[WID][LID] = arg[WID].res[off[WID] + LID];
        else
          v[WID][LID] = UINT32_MAX;
        __syncwarp();
        dup[WID][LID] = false;
        if (v[WID][LID] != UINT32_MAX)
          for (int l = 0; l < stk->level; ++l)
          {
            if (v[WID][LID] == cur_res[l])
            {
              dup[WID][LID] = true;
              break;
            }

            // dup[WID][LID] |= (v[WID][LID] == cur_res[v_order[l]]);
            // if (dup[WID][LID])
            //   break;
            // if (v[WID][LID] == cur_res[v_order[l]])
            // dup[WID][LID] = true;
            // if (v == stk->candidates_[l * C_NUM_CAN_UB + stk->iter[l]])
            // dup[WID][LID] = true;
          }
        __syncwarp();
        if (v[WID][LID] != UINT32_MAX &&
            !dup[WID][LID] &&
            (compact_encodings_[u * C_COL_LEN + v[WID][LID] / BLK_SIZE] & (1 << (v[WID][LID] % BLK_SIZE))))
        // if (v != UINT32_MAX && !dup[WID][LID] && (compact_encodings_[v * num_blocks + pos / BLK_SIZE] & (1 << (pos % BLK_SIZE))))
        {
          int mask = __activemask();
          int size = __popc(mask);
          int rank = __popc(mask & (FULL_MASK >> (31 - LID))) - 1;
          int my_pos = cnt[WID] + rank;
          // can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + my_pos] = v[WID][LID];
          stk->candidates_[stk->level * C_NUM_CAN_UB + my_pos] = v[WID][LID];
          if (rank == 0)
            cnt[WID] += size;
        }
        __syncwarp();
        if (LID == 0)
          off[WID] += warpSize;
        __syncwarp();
      }
      if (LID == 0)
      {
        *count += cnt[WID];
        stk->level--;
        stk->iter[stk->level]++;
        // cur_res[v_order[stk->level]] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
        // cur_res[order_obj->v_orders_[order_id * C_NUM_VQ + stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
        cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
        // cur_res[v_order[stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
      }
      __syncwarp();
    }
    else // intermediate levels
    {
      __shared__ int is_timeout[WARP_PER_BLOCK];
      // int is_timeout;
      if (LID == 0)
        is_timeout[WID] = stk->level == C_STOP_LEVEL - 1 && ELAPSED_TIME(start_clk) > TIMEOUT && !stk->stealed_task;
      __syncwarp();
      // is_timeout = __shfl_sync(FULL_MASK, is_timeout, 0);

      if (stk->num_candidates_[stk->level] == 0) // top-down, get new candidates and keep going down. Or return.
      {
        if (LID == 0)
          *ret = true;
        __syncwarp();
        get_new_v(
            d_degs_, d_offsets_, d_nbrs_,
            stk, cur, stealing_args, start_clk,
            order_obj, start_level,
            compact_encodings_, num_blocks,
            initial_task_table_, num_initial_task_table_rows,
            ret, cur_res, &s_order_id[WID]);
        if (*ret == false) // no candidates, return
        {
          if (LID == 0)
          {
            stk->iter[stk->level] = 0;
            stk->level--;
            stk->iter[stk->level]++;
            // cur_res[v_order[stk->level]] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
            // cur_res[order_obj->v_orders_[stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
            cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
          }
          __syncwarp();
        }
        else // candidates found, go down.
        {
          if (LID == 0)
          {
            stk->iter[stk->level] = 0;
            stk->level++;
            stk->iter[stk->level] = 0;
            stk->num_candidates_[stk->level] = 0;
          }
          __syncwarp();
        }
      }
      else if (stk->iter[stk->level] == stk->num_candidates_[stk->level]) // end of this level, return.
      {
        if (LID == 0)
        {
          stk->iter[stk->level] = 0;
          stk->num_candidates_[stk->level] = 0;

          stk->level--;
          stk->iter[stk->level]++;
          // cur_res[v_order[stk->level]] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
          // cur_res[v_order[stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
          // cur_res[order_obj->v_orders_[order_id * C_NUM_VQ + stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
          cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
        }
        __syncwarp();
        if (stk->level == start_level)
        {
          if (LID == 0)
            start_clk = clock64();
          __syncwarp();
          // start_clk = __shfl_sync(FULL_MASK, start_clk, 0);
        }
      }
      else // not end, map next v, then keep going down.
      {
        if (!is_timeout[WID])
        {
          if (LID == 0)
          {
            stk->level++;
            stk->iter[stk->level] = 0;
            stk->num_candidates_[stk->level] = 0;
          }
          __syncwarp();
        }
        else // timeout, split task into queue.
        {
          __shared__ bool enqueue_succ[WARP_PER_BLOCK];
          // int enqueue_succ = false;
          if (LID == 0)
            queue_arr_[0] = s_order_id[WID];
          else if (LID < C_STOP_LEVEL)
          {
            // queue_arr_[LID] = cur_res[v_order[LID]];
            queue_arr_[LID] = cur_res[LID - 1];
          }
          __syncwarp();
          if (LID == 0)
          {
            enqueue_succ[WID] = false;
            // for (int i = 0; i < C_STOP_LEVEL - 1; ++i)
            // {
            //   queue_arr_[i] = cur_res[v_order[i]];
            //   // queue_arr_[i] = stk->candidates_[i * C_NUM_CAN_UB + 0];

            //   // queue_arr_[i] = cur_res[v_order[i]];
            //   // if (stk->num_candidates_[i])
            //   //   // queue_arr_[i] = cur_res[v_order[i]];
            //   //   queue_arr_[i] = stk->candidates_[i * C_NUM_CAN_UB + stk->iter[i]];
            //   // else
            //   //   queue_arr_[i] = DeletionMarker<int>::val - 1;
            // }
            queue_arr_[C_STOP_LEVEL] = DeletionMarker<int>::val - 1;
            // queue_arr_[C_STOP_LEVEL - 1] = DeletionMarker<int>::val - 1;
#pragma unroll 2
            for (; stk->iter[stk->level] < stk->num_candidates_[stk->level]; ++stk->iter[stk->level])
            {
              // queue_arr_[C_STOP_LEVEL - 1] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
              // queue_arr_[C_STOP_LEVEL - 1] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
              queue_arr_[C_STOP_LEVEL] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
              enqueue_succ[WID] = stealing_args->queue->enqueue(queue_arr_, C_STOP_LEVEL + 1);
              if (!enqueue_succ[WID])
                break;
            }
          }
          __syncwarp();
          // enqueue_succ = __shfl_sync(FULL_MASK, enqueue_succ, 0);
          if (enqueue_succ[WID])
          {
            stk->num_candidates_[stk->level] = 0;
            stk->iter[stk->level] = 0;
            if (stk->level > start_level)
            {
              if (LID == 0)
              {
                stk->level--;
                stk->iter[stk->level]++;
                // cur_res[v_order[stk->level]] = can_wid[(stk->level - start_level - 1) * C_NUM_CAN_UB + stk->iter[stk->level]];
                cur_res[stk->level] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
                // cur_res[order_obj->v_orders_[order_id * C_NUM_VQ + stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
                // cur_res[v_order[stk->level]] = stk->candidates_[stk->level * C_NUM_CAN_UB + stk->iter[stk->level]];
              }
              __syncwarp();
            }
          }
          else
          {
            if (LID == 0)
              start_clk = clock64();
            __syncwarp();
            // start_clk = __shfl_sync(FULL_MASK, start_clk, 0);
          }
        }
      }
    }
  }
  __syncwarp();
}

__global__ void
parallel_match_kernel(
    degtype *__restrict__ d_degs_, offtype *__restrict__ d_offsets_, vtype *__restrict__ d_nbrs_,
    CallStack *__restrict__ call_stack, int *__restrict__ cur, unsigned long long *__restrict__ res,
    Queue *__restrict__ queue, OrderGPU *__restrict__ order_obj, int num_orders,

    int start_level,

    uint32_t *__restrict__ compact_encodings_,
    int num_blocks,
    vtype *__restrict__ d_u_candidate_vs_, numtype *__restrict__ d_num_u_candidate_vs_,

    vtype *__restrict__ initial_task_table_, numtype num_initial_task_table_rows)
{
  queue->init();

  // __shared__ OrderGPU s_order_obj(num_orders);
  __shared__ CallStack stk[WARP_PER_BLOCK];

  __shared__ unsigned long long s_count[WARP_PER_BLOCK]; // total count of results this warp
  __shared__ vtype cur_res[WARP_PER_BLOCK][MAX_VQ];      // current result, this warp
  // __shared__ vtype s_v_order[MAX_VQ];                    // shared across this block

  // for match()
  __shared__ bool ret[WARP_PER_BLOCK]; // return value of get_new_v(), this warp

  // for steal.queue
  __shared__ int queue_arr_[WARP_PER_BLOCK][MAX_VQ];
  __shared__ StealingArgs s_stealing_args;
  __shared__ long long start_clk[WARP_PER_BLOCK];

  s_stealing_args.queue = queue;

  // if (TID == 0)
  // {
  // s_order_obj = *order_obj;
  // }

  // if (TID < C_NUM_VQ)
  // {
  // s_v_order[TID] = s_order_obj.v_order_[TID];
  // }
  __syncthreads();

  if (LID == 0)
  {
    stk[WID].candidates_ = call_stack[WID_G].candidates_;
    // stk[WID] = call_stack[WID_G];
    stk[WID].level = start_level;
    stk[WID].num_candidates_[start_level] = 0;
    s_count[WID] = 0;
  }
  __syncwarp();

  // long long st = clock64();
  if (LID == 0)
    start_clk[WID] = clock64();
  __syncwarp();
  match(
      d_degs_, d_offsets_, d_nbrs_,
      &stk[WID], cur,
      &s_count[WID], &s_stealing_args,
      start_clk[WID], order_obj,

      start_level, queue_arr_[WID],

      compact_encodings_,
      num_blocks,

      initial_task_table_, num_initial_task_table_rows,

      &ret[WID], cur_res[WID]);
  __syncwarp();

  // long long ed = clock64();

  if (LID == 31)
  {
    res[WID_G] = s_count[WID];
    // printf("bid: %d, WID: %d, time: %lf(ms)\n", BID, WID_G, (ed - start_clk) / (1.0 * CLOCK_RATE));
    // printf("WID: %d, count: %lu\n", WID_G, s_count[WID]);
  }
  __syncwarp();
}

__global__ void
vertexJoinBFS(
    offtype *d_offsets_, vtype *d_neighbors_, degtype *d_degree_,
    OrderGPU *order_obj,
    int level, // size of one row = level + 1(order_id)  ,, to be mapped level-th u.
    vtype *intersect_temp_storage, numtype *num_intersect_temp_storage,

    uint32_t *d_encodings_, numtype num_blocks, int *d_enc_pos_u_,
    vtype *d_res_table_old, numtype num_res_old,
    vtype *d_res_table, numtype *num_res_new,
    int *exceed)
{
  // int tid = threadIdx.x;
  // int bid = blockIdx.x;
  // int idx = tid + bid * blockDim.x;
  // int wid = tid >> 5;
  // int lid = tid & 31;
  // int wid_g = idx >> 5;

  __shared__ int s_row[WARP_PER_BLOCK];
  __shared__ vtype s_cur_res[WARP_PER_BLOCK][MAX_VQ];
  __shared__ vtype mapped_vs_[WARP_PER_BLOCK][MAX_VQ];
  __shared__ Arg_t arg[WARP_PER_BLOCK];
  __shared__ int warp_pos[WARP_PER_BLOCK];
  // __shared__ int s_v_order_[MAX_VQ];
  // __shared__ int s_num_bn[MAX_VQ];
  // __shared__ vtype s_bn[MAX_VQ][MAX_VQ];

  __shared__ int s_order_id[WARP_PER_BLOCK];
  __shared__ vtype s_u[WARP_PER_BLOCK];
  __shared__ int num_warps;
  __shared__ int block_iter_cnt[WARP_PER_BLOCK];
  // __shared__ int s_pos_u[MAX_VQ];
  // __shared__ int s_orders_[32][MAX_VQ];

  __shared__ int min_i[WARP_PER_BLOCK];
  __shared__ int min_nbrs[WARP_PER_BLOCK];
  __shared__ bool flag[WARP_PER_BLOCK][MAX_VQ];

  // if (LID < C_NUM_VQ)
  //   flag[WID][LID] = false;
  // __syncwarp();

  // int block_iter_cnt = 0;

  if (LID == 0)
    block_iter_cnt[WID] = 0;
  __syncwarp();

  if (TID == 0)
    num_warps = blockDim.x / warpSize * gridDim.x;
  __syncthreads();

  if (TID < C_NUM_VQ)
  {
    // s_pos_u[TID] = d_enc_pos_u_[TID];
    // s_num_bn[TID] = order_obj->num_backward_neighbors_[TID];
    // s_v_order_[TID] = order_obj->v_orders_[TID];
  }
  __syncthreads();
  // if (TID < C_NUM_VQ * C_NUM_VQ)
  //   s_bn[TID / C_NUM_VQ][TID % C_NUM_VQ] = order_obj->backward_neighbors_[TID];
  // __syncthreads();

  // vtype u = s_v_order_[level];

  while (WID_G + block_iter_cnt[WID] * num_warps < num_res_old)
  {
    if (LID < C_NUM_VQ)
      flag[WID][LID] = false;
    __syncwarp();
    if (LID == 0)
    {
      s_row[WID] = WID_G + block_iter_cnt[WID] * num_warps;
      s_order_id[WID] = d_res_table_old[s_row[WID] * (level + 1)];
      s_u[WID] = order_obj->v_orders_[s_order_id[WID] * C_NUM_VQ + level];
    }
    __syncwarp();
    if (LID < level)
      s_cur_res[WID][LID] = d_res_table_old[s_row[WID] * (level + 1) + 1 + LID];
    __syncwarp();

    // if (LID < s_num_bn[u])
    if (LID < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]])
    {
      vtype u_back = order_obj->backward_neighbors_[s_order_id[WID] * C_NUM_VQ * C_NUM_VQ + s_u[WID] * C_NUM_VQ + LID];
      mapped_vs_[WID][LID] = s_cur_res[WID][order_obj->u2ls_[s_order_id[WID] * C_NUM_VQ + u_back]];
      // vtype v_back = s_cur_res[WID][u_back];
      // mapped_vs_[WID][LID] = v_back;
    }
    __syncwarp();

    if (LID == 0)
    {
      min_i[WID] = 0;
      // if (order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + level] == 1)
      // {
      // min_i[WID] = 0;
      // }
      if (order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]] > 1)
      {
        // min_i[WID] = 0;
        min_nbrs[WID] = d_degree_[mapped_vs_[WID][0]];
        for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]]; ++i)
        {
          if (d_degree_[mapped_vs_[WID][i]] < min_nbrs[WID])
          {
            min_i[WID] = i;
            min_nbrs[WID] = d_degree_[mapped_vs_[WID][i]];
          }
        }
      }
    }
    __syncwarp();

    if (LID == 0)
    {
      // printf("mapped_vs_[0] = %d\n", mapped_vs_[0]);
      arg[WID].res = d_neighbors_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
      arg[WID].res_size = num_intersect_temp_storage + WID_G;
      arg[WID].res_size[0] = d_degree_[mapped_vs_[WID][min_i[WID]]];
      flag[WID][min_i[WID]] = true;
    }
    __syncwarp();
    for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]]; ++i)
    {
      if (LID == 0)
      {
        min_nbrs[WID] = UINT32_MAX;
        for (int j = 0; j < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + s_u[WID]]; ++j)
        {
          if (flag[WID][j])
            continue;
          if (d_degree_[mapped_vs_[WID][j]] < min_nbrs[WID])
          {
            min_i[WID] = j;
            min_nbrs[WID] = d_degree_[mapped_vs_[WID][j]];
          }
        }

        arg[WID].set1 = arg[WID].res;
        arg[WID].set1_size = arg[WID].res_size[0];
        arg[WID].set2 = d_neighbors_ + d_offsets_[mapped_vs_[WID][min_i[WID]]];
        arg[WID].set2_size = d_degree_[mapped_vs_[WID][min_i[WID]]];
        // arg[WID].res = can + (stk->level - start_level - 1) * C_NUM_CAN_UB;
        arg[WID].res = intersect_temp_storage + WID_G * C_NUM_CAN_UB;
        arg[WID].res_size = num_intersect_temp_storage + WID_G;
        flag[WID][min_i[WID]] = true;
      }
      __syncwarp();
      intersect(&arg[WID]);
    }

    // if (LID == 0)
    // {
    //   arg[WID].res = d_neighbors_ + d_offsets_[mapped_vs_[WID][0]];
    //   arg[WID].res_size = num_intersect_temp_storage + WID_G;
    //   arg[WID].res_size[0] = d_degree_[mapped_vs_[WID][0]];
    // }
    // __syncwarp();
    // for (int i = 1; i < order_obj->num_backward_neighbors_[s_order_id[WID] * C_NUM_VQ + level]; ++i)
    // // for (int i = 1; i < s_num_bn[u]; ++i)
    // {
    //   if (LID == 0)
    //   {
    //     arg[WID].set1 = arg[WID].res;
    //     arg[WID].set1_size = arg[WID].res_size[0];
    //     arg[WID].set2 = d_neighbors_ + d_offsets_[mapped_vs_[WID][i]];
    //     arg[WID].set2_size = d_degree_[mapped_vs_[WID][i]];
    //     arg[WID].res = intersect_temp_storage + WID_G * C_NUM_CAN_UB;
    //     arg[WID].res_size = num_intersect_temp_storage + WID_G;
    //   }
    //   __syncwarp();
    //   intersect(&arg[WID]);
    // }
    // int pos = s_pos_u[s_u[WID]];

    offtype off = 0;
    offtype off_ed = arg[WID].res_size[0];
    vtype v;
    bool dup = false;
    while (off < off_ed)
    {
      int my_off = off + LID;
      if (my_off < off_ed)
        v = arg[WID].res[my_off];
      else
        v = UINT32_MAX;
      __syncwarp();
      dup = false;
      if (v != UINT32_MAX)
        for (int l = 0; l < level; ++l)
        {
          // if (v == s_cur_res[WID][order_obj->v_orders_[s_order_id[WID] * C_NUM_VQ + l]])
          // if (v == s_cur_res[WID][s_v_order_[l]])
          if (v == s_cur_res[WID][l])
          {
            dup = true;
            break;
          }
        }
      __syncwarp();
      if (v != UINT32_MAX &&
          !dup &&
          // (d_encodings_[v * num_blocks + s_pos_u[s_u[WID]] / BLK_SIZE] & (1 << (s_pos_u[s_u[WID]] % BLK_SIZE)))
          d_encodings_[s_u[WID] * C_COL_LEN + v / BLK_SIZE] & (1 << (v % BLK_SIZE)))
      {
        int mask = __activemask();
        int size = __popc(mask);
        int rank = __popc(mask & (FULL_MASK >> (31 - LID))) - 1;
        if (rank == 0)
        {
          warp_pos[WID] = atomicAdd(num_res_new, size);
          if ((warp_pos[WID] + size) * (level + 2) >= (MAX_RES)*C_NUM_VQ)
          {
            *exceed = 1;
          }
        }
        __syncwarp(mask);
        // if (*exceed == 1)
        //   return;
        if (*exceed == 0)
        {
          int my_pos = warp_pos[WID] + rank;

          d_res_table[my_pos * (level + 2)] = s_order_id[WID];
          for (int i = 0; i < level; ++i)
            d_res_table[my_pos * (level + 2) + 1 + i] = s_cur_res[WID][i];
          d_res_table[my_pos * (level + 2) + 1 + level] = v;
        }
        // for (int i = 0; i < C_NUM_VQ; ++i)
        // d_res_table[my_pos * C_NUM_VQ + i] = d_res_table_old[s_row[WID] * C_NUM_VQ + i];
        // d_res_table[my_pos * C_NUM_VQ + s_u[WID]] = v;
      }
      __syncwarp();
      if (*exceed == 1)
        return;
      off += warpSize;
    }
    if (LID == 0)
      block_iter_cnt[WID]++;
    __syncwarp();
  }
}

void parallelMatch(
    cpuGraph *hq, cpuGraph *hg,
    gpuGraph *dq, gpuGraph *dg,
    OrderCPU *h_order_obj,

    uint32_t *d_compact_encodings_,
    encodingMeta *enc_meta,
    uint32_t *d_u_candidate_vs_, numtype *d_num_u_candidate_vs_,
    numtype *h_num_u_candidate_vs_,

    ResTable *res_table)
{
  // gpuGraph *real_dg;
  // cuchk(hipMalloc((void **)&real_dq, sizeof(gpuGraph)));
  // cuchk(hipMalloc((void **)&real_dg, sizeof(gpuGraph)));

  // cuchk(hipMemcpy(real_dq, dq, sizeof(gpuGraph), hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(real_dg, dg, sizeof(gpuGraph), hipMemcpyHostToDevice));

  // std::cout << "free memory: " << getFreeGlobalMemory(GPU_NUM) << std::endl;
  // std::cout << "MAX_L_FREQ: " << MAX_L_FREQ << std::endl;
  // std::cout << "NUM_VQ: " << NUM_VQ << std::endl;

  NUM_CAN_UB = 0;
  for (int i = 0; i < NUM_VQ; ++i)
    NUM_CAN_UB = std::max(NUM_CAN_UB, h_num_u_candidate_vs_[i]);
  NUM_CAN_UB = std::min(NUM_CAN_UB, MAX_DATA_DEGREE);
  hipMemcpyToSymbol(HIP_SYMBOL(C_NUM_CAN_UB), &NUM_CAN_UB, sizeof(uint32_t));
  // std::cout << "NUM_CAN_UB: " << NUM_CAN_UB << std::endl;

  /*--- tdfs ---*/
  CallStack *callstack_gpu;
  std::vector<CallStack> stk(NWARPS_TOTAL);

  vtype *candidate_space;
  cuchk(hipMalloc((void **)&candidate_space, sizeof(vtype) * NUM_VQ * NUM_CAN_UB * NWARPS_TOTAL));

  for (int i = 0; i < NWARPS_TOTAL; i++)
  {
    auto &s = stk[i];
    s.candidates_ = candidate_space + i * NUM_CAN_UB * NUM_VQ;
    memset(s.iter, 0, sizeof(vtype) * MAX_VQ);
    memset(s.num_candidates_, 0, sizeof(numtype) * MAX_VQ);
    // memset(s.map_res_, 0, sizeof(s.map_res_));
  }
  cuchk(hipMalloc(&callstack_gpu, NWARPS_TOTAL * sizeof(CallStack)));
  cuchk(hipMemcpy(callstack_gpu, stk.data(), sizeof(CallStack) * NWARPS_TOTAL, hipMemcpyHostToDevice));

  int *cur;
  cuchk(hipMalloc((void **)&cur, sizeof(int)));
  cuchk(hipMemset(cur, 0, sizeof(int)));

  unsigned long long *d_res;
  cuchk(hipMalloc((void **)&d_res, sizeof(unsigned long long) * NWARPS_TOTAL));
  cuchk(hipMemset(d_res, 0, sizeof(unsigned long long) * NWARPS_TOTAL));

  numtype &num_orders = h_order_obj->num_orders;
  OrderGPU order_gpu_temp(num_orders);
  cuchk(hipMemcpy(order_gpu_temp.num_orders, &num_orders, sizeof(numtype), hipMemcpyHostToDevice));
  // order_gpu_temp.num_orders = num_orders;
  cuchk(hipMemcpy(order_gpu_temp.roots_, h_order_obj->roots.data(), sizeof(vtype) * num_orders, hipMemcpyHostToDevice));
  for (int i = 0; i < num_orders; ++i)
  {
    cuchk(hipMemcpy(order_gpu_temp.v_orders_ + i * NUM_VQ, h_order_obj->v_orders[i].data(), sizeof(vtype) * NUM_VQ, hipMemcpyHostToDevice));
    cuchk(hipMemcpy(order_gpu_temp.u2ls_ + i * NUM_VQ, h_order_obj->u2ls[i].data(), sizeof(int) * NUM_VQ, hipMemcpyHostToDevice));
    cuchk(hipMemcpy(order_gpu_temp.num_backward_neighbors_ + i * NUM_VQ, h_order_obj->num_backward_neighbors[i].data(), sizeof(numtype) * NUM_VQ, hipMemcpyHostToDevice));
  }
  // cuchk(hipMemcpy(order_gpu_temp.v_orders_, h_order_obj->v_orders.data(), sizeof(vtype) * num_orders * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.u2ls_, h_order_obj->u2ls.data(), sizeof(int) * num_orders * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.num_backward_neighbors_, h_order_obj->num_backward_neighbors.data(), sizeof(numtype) * num_orders * NUM_VQ, hipMemcpyHostToDevice));
  offtype off = 0;
  for (int i = 0; i < num_orders; ++i)
  {
    for (int j = 0; j < NUM_VQ; ++j)
    {
      cuchk(hipMemcpy(order_gpu_temp.backward_neighbors_ + off, h_order_obj->backward_neighbors[i][j].data(), sizeof(vtype) * h_order_obj->num_backward_neighbors[i][j], hipMemcpyHostToDevice));
      off += NUM_VQ;
    }
  }

  // cuchk(hipMemcpy(order_gpu_temp.root_u, &h_order_obj->root_u, sizeof(vtype), hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.v_order_, h_order_obj->v_order_, sizeof(vtype) * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.u2l_, h_order_obj->u2l_, sizeof(int) * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.e_order_, h_order_obj->e_order_, sizeof(etype) * NUM_EQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.shared_neighbors_with_, h_order_obj->shared_neighbors_with_, sizeof(vtype) * NUM_VQ, hipMemcpyHostToDevice));
  // cuchk(hipMemcpy(order_gpu_temp.num_backward_neighbors_, h_order_obj->num_backward_neighbors_, sizeof(numtype) * NUM_VQ, hipMemcpyHostToDevice));
  // offtype off = 0;
  // for (int i = 0; i < NUM_VQ; ++i)
  // {
  // cuchk(hipMemcpy(order_gpu_temp.backward_neighbors_ + off, h_order_obj->backward_neighbors_[i], sizeof(vtype) * NUM_VQ, hipMemcpyHostToDevice));
  // off += NUM_VQ;
  // }

  /*--- order, encoding ---*/
  OrderGPU *real_order_gpu;
  cuchk(hipMalloc((void **)&real_order_gpu, sizeof(OrderGPU)));
  cuchk(hipMemcpy(real_order_gpu, &order_gpu_temp, sizeof(OrderGPU), hipMemcpyHostToDevice));

  numtype &num_blocks = enc_meta->num_blocks;

  // vtype u = h_order_obj->v_order_[0];

  vtype *d_res_table_old_;
  // unsigned long long h_num_res_old = h_num_u_candidate_vs_[u];
  unsigned long long h_num_res_old;
  cuchk(hipMalloc((void **)&d_res_table_old_, sizeof(vtype) * NUM_VQ * MAX_RES));

  vtype *d_res_table_;
  unsigned long long h_num_res_new = 0;
  cuchk(hipMalloc((void **)&d_res_table_, sizeof(vtype) * NUM_VQ * MAX_RES));

  int start_level = 0; // at `start_level`, tasks are done, just fetch from table. real match begins from `start_level + 1`.

  vtype *d_intersect_temp_storage;
  cuchk(hipMalloc((void **)&d_intersect_temp_storage, sizeof(vtype) * NUM_CAN_UB * NWARPS_TOTAL));
  numtype *d_num_intersect_temp_storage;
  cuchk(hipMalloc((void **)&d_num_intersect_temp_storage, sizeof(numtype) * NWARPS_TOTAL));
  cuchk(hipMemset(d_num_intersect_temp_storage, 0, sizeof(numtype) * NWARPS_TOTAL));

  warmup<<<GRID_DIM, BLOCK_DIM>>>();
  cuchk(hipDeviceSynchronize());

  TIME_INIT();
  TIME_START();

  // struct timespec time_st;
  // struct timespec time_ed;

  micro_init();
  micro_start();

  // clock_gettime(CLOCK_REALTIME, &time_st);
  // dim3 fj_block = BLOCK_DIM;
  // dim3 fj_grid = (h_num_res_old - 1) / fj_block.x + 1;
  // firstJoinKernel<<<fj_grid, fj_block>>>(u, d_u_candidate_vs_, h_num_u_candidate_vs_[u], d_res_table_old_);
  // cuchk(hipDeviceSynchronize());
  // std::cout << "first join done" << std::endl;

  // hipFree(d_u_candidate_vs_);
  // hipFree(d_num_u_candidate_vs_);

  int level = 2;
  cuchk(hipMemcpy(d_res_table_old_, res_table->res_table, sizeof(vtype) * res_table->size, hipMemcpyHostToDevice));
  h_num_res_old = res_table->size / (level + 1);

  // #ifndef NDEBUG
  // std::cout << "res_table: " << std::endl;
  // for (int i = 0; i < res_table->size; ++i)
  // {
  //   std::cout << res_table->res_table[i] << " ";
  // }
  // #endif

  // offtype e_off;
  // for (e_off = 0; e_off < NUM_EQ; ++e_off)
  // {
  //   if (h_order_obj->e_is_tree_[e_off] == true) // if not a tree-edge, no need to break.
  //     if (h_num_res_old > MAX_RES / 100)
  //     // if (h_num_res_old > 1880000)
  //     {
  //       // std::cout << "e_off = " << e_off << " break" << std::endl;
  //       break;
  //     }

  //   etype e = h_order_obj->e_order_[e_off];
  //   vtype u = hq->evv[e].second;
  //   vtype u_matched = hq->evv[e].first;

  //   if (h_order_obj->e_is_tree_[e_off])
  //   {
  //     start_level++;
  //     joinOneEdge(
  //         hq, hg, dq, dg, u, u_matched,
  //         d_res_table_old_, h_num_res_old,
  //         d_res_table_, h_num_res_new,
  //         d_compact_encodings_,
  //         enc_meta);
  //   }
  //   else
  //   {
  //     unsigned long long *d_num_res_new;
  //     hipMalloc((void **)&d_num_res_new, sizeof(unsigned long long));
  //     hipMemset(d_num_res_new, 0, sizeof(unsigned long long));

  //     int enc_pos_u = -1, enc_pos_u_matched = -1;

  //     dim3 spm_block = BLOCK_DIM;
  //     int N = h_num_res_old;
  //     dim3 spm_grid = std::min(GRID_DIM, calc_grid_dim(N, spm_block.x));
  //     selectPartialMatchingsKernel<<<spm_grid, spm_block>>>(
  //         dg->offsets_, dg->neighbors_,
  //         u, u_matched,
  //         d_res_table_old_, h_num_res_old,
  //         d_res_table_, d_num_res_new);
  //     cuchk(hipDeviceSynchronize());
  //     cuchk(hipMemcpy(&h_num_res_new, d_num_res_new, sizeof(unsigned long long), hipMemcpyDeviceToHost));
  //     // std::cout << "select, res: " << h_num_res_new << std::endl;
  //   }
  //   std::swap(d_res_table_old_, d_res_table_);
  //   h_num_res_old = h_num_res_new;
  //   h_num_res_new = 0;
  // }

  offtype v_off = 2; // to be mapped

  int *d_enc_pos_u_;
  // cuchk(hipMalloc((void **)&d_enc_pos_u_, sizeof(int) * NUM_VQ));
  // cuchk(hipMemcpy(d_enc_pos_u_, enc_meta->enc_pos_of_u_, sizeof(int) * NUM_VQ, hipMemcpyHostToDevice));

  numtype *d_num_res_new;
  cuchk(hipMalloc((void **)&d_num_res_new, sizeof(numtype)));
  cuchk(hipMemset(d_num_res_new, 0, sizeof(numtype)));

  int *exceed;
  cuchk(hipMalloc((void **)&exceed, sizeof(int)));
  cuchk(hipMemset(exceed, 0, sizeof(int)));

  int h_exceed;
  while (v_off < NUM_VQ)
  {
    vertexJoinBFS<<<GRID_DIM, BLOCK_DIM>>>(
        dg->offsets_, dg->neighbors_, dg->degree_,
        real_order_gpu,
        v_off,
        d_intersect_temp_storage, d_num_intersect_temp_storage,

        d_compact_encodings_, num_blocks, d_enc_pos_u_,
        d_res_table_old_, h_num_res_old,
        d_res_table_, d_num_res_new,
        exceed);
    cuchk(hipDeviceSynchronize());

    cuchk(hipMemcpy(&h_exceed, exceed, sizeof(int), hipMemcpyDeviceToHost));
    if (h_exceed == 1)
    {
      // std::cout << "exceed" << std::endl;
      break;
    }

    // auto num_res_backup = h_num_res_old;
    cuchk(hipMemcpy(&h_num_res_old, d_num_res_new, sizeof(numtype), hipMemcpyDeviceToHost));
    std::swap(d_res_table_old_, d_res_table_);
    cuchk(hipMemset(d_num_res_new, 0, sizeof(numtype)));

    v_off++;
  }

  micro_end();
  // micro_print_local("bfs");
  micro_calc_local();
  total_match_time_us += diff_micro.tv_nsec / 1000.0;

  TIME_END();
  total_match_time_ms += kernel_time;

  // clock_gettime(CLOCK_REALTIME, &time_ed);

  // std::cout << "bfs: " << time_ed.tv_nsec - time_st.tv_nsec << "(ns)" << std::endl;
  // TIME_END();
  // PRINT_LOCAL_TIME("BFS");

  cuchk(hipFree(d_res_table_));
  if (v_off == NUM_VQ)
  {
    // std::cout << "res: " << h_num_res_old << std::endl;
    // std::cout << std::endl;
    return;
  }
  // std::cout << "u: " << h_order_obj->v_orders[start_level] << " num_rows: " << h_num_res_old << std::endl;

#ifndef NDEBUG
  std::cout << "prepare done, entering match kernel" << std::endl;
#endif

  start_level = v_off - 1;
  if (start_level >= NUM_VQ - 1)
  {
    // std::cout << "res: " << h_num_res_old << std::endl;
    return;
  }

  // STOP_LEVEL = std::max((uint32_t)start_level + 1, NUM_VQ / 2);
  // start_level--;
  STOP_LEVEL = std::min((unsigned)start_level + 2, NUM_VQ - 1);
  cuchk(hipMemcpyToSymbol(HIP_SYMBOL(C_STOP_LEVEL), &STOP_LEVEL, sizeof(uint32_t)));

  int *gpu_timeout_queue_space;
  cuchk(hipMalloc(&gpu_timeout_queue_space, sizeof(int) * TIMEOUT_QUEUE_CAP * STOP_LEVEL));
  // vtype *gpu_timeout_candidate_queue_space;
  // hipMalloc(&gpu_timeout_candidate_queue_space, sizeof(vtype) * TIMEOUT_QUEUE_CAP * WARP_SIZE);
  // hipMemset(gpu_timeout_candidate_queue_space, UINT32_MAX, sizeof(vtype) * TIMEOUT_QUEUE_CAP * WARP_SIZE);
  Queue *gpu_timeout_queue;
  hipMallocManaged(&gpu_timeout_queue, sizeof(Queue));
  gpu_timeout_queue->queue_ = gpu_timeout_queue_space;
  // gpu_timeout_queue->candidate_queue = gpu_timeout_candidate_queue_space;
  gpu_timeout_queue->size_ = TIMEOUT_QUEUE_CAP * (STOP_LEVEL);
  gpu_timeout_queue->resetQueue();
  hipMemPrefetchAsync(gpu_timeout_queue, sizeof(Queue), GPU_NUM);
  hipDeviceSynchronize();

  // size_t shared_used = 0;
  // shared_used += sizeof(OrderGPU);
  // shared_used += sizeof(gpuGraph);
  // shared_used += sizeof(CallStack) * WARP_PER_BLOCK;
  // shared_used += sizeof(unsigned long long) * WARP_PER_BLOCK;
  // shared_used += sizeof(int) * MAX_VQ;
  // shared_used += sizeof(vtype) * WARP_PER_BLOCK * MAX_VQ;
  // shared_used += sizeof(vtype) * MAX_VQ;
  // shared_used += sizeof(Arg_t) * WARP_PER_BLOCK;
  // shared_used += sizeof(bool) * WARP_PER_BLOCK;
  // shared_used += sizeof(vtype) * WARP_PER_BLOCK * MAX_VQ;
  // shared_used += sizeof(int) * WARP_PER_BLOCK;
  // shared_used += sizeof(int) * WARP_PER_BLOCK * MAX_VQ;
  // shared_used += sizeof(StealingArgs);
  // std::cout << shared_used * 1.0 / 1024 << "KB" << std::endl;
  // std::cout << "totally: " << shared_used * 1.0 / 1024 * GRID_DIM << "KB" << std::endl;

  // TIME_INIT();
  TIME_START();
  micro_start();

  // clock_gettime(CLOCK_REALTIME, &time_st);

  // std::cout << "st_level = " << start_level << std::endl;
  // std::cout << "num_rows = " << h_num_res_old << std::endl;

  parallel_match_kernel<<<GRID_DIM, BLOCK_DIM>>>(
      dg->degree_, dg->offsets_, dg->neighbors_,
      callstack_gpu, cur, d_res,
      gpu_timeout_queue, real_order_gpu, h_order_obj->num_orders,

      start_level,

      d_compact_encodings_,
      enc_meta->num_blocks,
      d_u_candidate_vs_, d_num_u_candidate_vs_,

      d_res_table_old_, h_num_res_old);
  cuchk(hipDeviceSynchronize());

  micro_end();
  // micro_print_local("dfs");
  micro_calc_local();
  total_match_time_us += diff_micro.tv_nsec / 1000.0;
  // clock_gettime(CLOCK_REALTIME, &time_ed);

  // std::cout << "dfs: " << time_ed.tv_nsec - time_st.tv_nsec << "(ns)" << std::endl;

  TIME_END();
  // PRINT_LOCAL_TIME("DFS_JOIN");
  total_match_time_ms += kernel_time;
  // PRINT_TOTAL_TIME("HYBRID_JOIN");
  // std::cout << "Parallel_match_kernel done" << std::endl;

  unsigned long long *h_res = new unsigned long long[NWARPS_TOTAL];
  cuchk(hipMemcpy(h_res, d_res, sizeof(unsigned long long) * NWARPS_TOTAL, hipMemcpyDeviceToHost));
  unsigned long long res = 0;
  for (int i = 0; i < NWARPS_TOTAL; ++i)
    res += h_res[i];
  // std::cout << "res: " << res << std::endl;
  // std::cout << std::endl;

  delete[] h_res;

  cuchk(hipFree(d_res_table_old_));
  cuchk(hipFree(d_res));
  cuchk(hipFree(cur));
  cuchk(hipFree(callstack_gpu));
  cuchk(hipFree(candidate_space));
  cuchk(hipFree(d_intersect_temp_storage));
  cuchk(hipFree(d_num_intersect_temp_storage));
  cuchk(hipFree(d_num_res_new));
  cuchk(hipFree(exceed));
  cuchk(hipFree(gpu_timeout_queue_space));
  cuchk(hipFree(real_order_gpu));
  // cuchk(hipFree(real_dg));
  // cuchk(hipFree(real_dq));
  // cuchk(hipFree(d_enc_pos_u_));
  // cuchk(hipFree(gpu_timeout_candidate_queue_space));
  cuchk(hipFree(gpu_timeout_queue));
}